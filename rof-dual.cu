#include "hip/hip_runtime.h"
#include <algorithm>
#include <stdio.h>
#include <math.h>
#include <omp.h>
#include "utils.h"
#include <stdlib.h>

struct RGBImage {
  long Xsize;
  long Ysize;
  float* A;
};

/*
 sample from normal dist
*/
float randn (float mu, float sigma) {
  float U1, U2, W, mult;
  static float X1, X2;
  static int call = 0;
 
  if (call == 1)
    {
      call = !call;
      return (mu + sigma * (float) X2);
    }
 
  do
    {
      U1 = -1 + ((float) rand () / RAND_MAX) * 2;
      U2 = -1 + ((float) rand () / RAND_MAX) * 2;
      W = pow (U1, 2) + pow (U2, 2);
    }
  while (W >= 1 || W == 0);
 
  mult = sqrt ((-2 * log (W)) / W);
  X1 = U1 * mult;
  X2 = U2 * mult;
 
  call = !call;
 
  return (mu + sigma * (float) X1);
}

void read_image(const char* fname, RGBImage* I) {
  I->Xsize = 0;
  I->Ysize = 0;
  I->A = NULL;

  FILE* f = fopen(fname, "rb");
  if (f == NULL) return;
  fscanf(f, "P6\n%d %d\n255\n", &I->Ysize, &I->Xsize);
  long N = I->Xsize * I->Ysize;
  if (N) {
    I->A = (float*) malloc(3*N * sizeof(float));
    unsigned char* I0 = (unsigned char*) malloc(3*N * sizeof(unsigned char));
    fread(I0, sizeof(unsigned char), 3*N, f);
    for (long i0 = 0; i0 < N; i0++) {
      for (long i1 = 0; i1 < 3; i1++) {
        I->A[i1*N+i0] = I0[i0*3+i1];
      }
    }
    free(I0);
  }
  fclose(f);
}

void write_image(const char* fname, const RGBImage I) {
  long N = I.Xsize * I.Ysize;
  if (!N) return;

  FILE* f = fopen(fname, "wb");
  if (f == NULL) return;
  fprintf(f, "P6\n%d %d\n255\n", I.Ysize, I.Xsize);
  unsigned char* I0 = (unsigned char*) malloc(3*N * sizeof(unsigned char));
  for (long i0 = 0; i0 < 3; i0++) {
    for (long i1 = 0; i1 < N; i1++) {
      I0[i1*3+i0] = I.A[i0*N+i1];
    }
  }
  fwrite(I0, sizeof(unsigned char), 3*N, f);
  free(I0);
  fclose(f);
}

void free_image(RGBImage* I) {
  long N = I->Xsize * I->Ysize;
  if (N) free(I->A);
  I->A = NULL;
}

#define BLOCK_DIM 32

/*
ROF-dual kernel without using shared memory
*/
__global__ void rof(float* u, float* p0x, float* p1x, float* p0y, float* p1y, float* f, float* gradx, float* grady,  float lambda, float tau, long Xsize, long Ysize, float h, float* div){
  int idx = (blockIdx.x)*BLOCK_DIM + threadIdx.x;
  int idy = (blockIdx.y)*BLOCK_DIM + threadIdx.y;
  // u = f + lambda div(p)
  if (idx < Xsize && idy < Ysize) {
    u[idx*Ysize + idy] = f[idx*Ysize + idy] + lambda*div[idx*Ysize+idy];
  }
  __syncthreads();
  // gradient of u
  if (idx < Xsize-1 && idy < Ysize-1) {
    gradx[idx*Ysize+idy] = (u[(idx+1)*Ysize + idy] - u[idx*Ysize + idy]);
    grady[idx*Ysize+idy] = (u[idx*Ysize + idy+1] - u[idx*Ysize + idy]);  
  }
  // gradient descent step
  if (idx < Xsize && idy < Ysize) {
    float numx = p0x[idx*Ysize+idy] + (tau/lambda)*gradx[idx*Ysize+idy];
    float numy = p0y[idx*Ysize+idy] + (tau/lambda)*grady[idx*Ysize+idy];
    float norm = sqrt( numx*numx + numy*numy);
    p1x[idx*Ysize + idy] = numx/max(1.0,norm); 
    p1y[idx*Ysize+idy] = numy/max(1.0,norm);
  }
  __syncthreads();
  float ux;
  float uy;
  // derivatives of p for boundaries
  if ( idx == 0) {
    ux = p1x[idx*Ysize + idy];
  }  
  if (idx == Xsize -1 ) {
    ux = -p1x[(idx-1)*Ysize + idy];
  }
  if (idy == 0){
    uy = p1y[idx*Ysize + idy];
  }
  if (idy == Ysize-1){
    uy = -p1y[idx*Ysize + idy-1];
  }
  // derivatives of p for non-boundaries
  if (idx > 0 && idx < Xsize -1) {
    ux = p1x[idx*Ysize + idy] - p1x[(idx-1)*Ysize + idy];
  }
  if (idy > 0 && idy < Ysize -1) {
    uy = p1y[idx*Ysize + idy] - p1y[idx*Ysize + idy-1];
  }
  // div p
  if (idx < Xsize && idy < Ysize) {
    div[idx*Ysize + idy] = ux + uy;
    p0x[idx*Ysize + idy] = p1x[idx*Ysize + idy];
    p0y[idx*Ysize + idy] = p1y[idx*Ysize + idy];
  }
}

/*
ROF-dual with shared mem for gradx, grady
*/
__global__ void rof_gsmem(float* px, float* py, float* f, float lambda, float tau, long Xsize, long Ysize, float* div){
  int idx = (blockIdx.x)*BLOCK_DIM + threadIdx.x;
  int idy = (blockIdx.y)*BLOCK_DIM + threadIdx.y;
  __shared__ float gradx[BLOCK_DIM+1][BLOCK_DIM+1]; // gradient of u in x direction
  __shared__ float grady[BLOCK_DIM+1][BLOCK_DIM+1]; // gradient of u in y direction
  __shared__ float u[BLOCK_DIM+2][BLOCK_DIM+2];
  __shared__ float pxsh[BLOCK_DIM+1][BLOCK_DIM+1]; // x coordinate of vector p
  __shared__ float pysh[BLOCK_DIM+1][BLOCK_DIM+1]; // y coordinate of vector p
  float numx, numy, norm;

  // u = f + lambda div p for ghost cells of shared mem block 
  if(blockIdx.x < Xsize/BLOCK_DIM && threadIdx.x == BLOCK_DIM -1){
    u[BLOCK_DIM+1][threadIdx.y+1] = f[(idx+1)*Ysize + idy] + lambda*div[(idx+1)*Ysize+idy];
  }
  if(blockIdx.y < Ysize/BLOCK_DIM && threadIdx.y == BLOCK_DIM -1){
    u[threadIdx.x+1][BLOCK_DIM+1] = f[idx*Ysize + idy + 1] + lambda*div[idx*Ysize+idy+1];;
  }
  if(blockIdx.x > 0 && threadIdx.x == 0) {
    u[0][threadIdx.y+1] = f[ (idx-1)*Ysize + idy] + lambda*div[(idx-1)*Ysize + idy];
  }
  if(blockIdx.y > 0 && threadIdx.y == 0){
    u[threadIdx.x+1][0] = f[idx*Ysize + idy-1] + lambda*div[idx*Ysize + idy-1];
  }

  // u = f + lambda div p
  if (idx < Xsize && idy < Ysize) {
    u[threadIdx.x+1][threadIdx.y+1] = f[idx*Ysize + idy] + lambda*div[idx*Ysize+idy];
  }
  __syncthreads();
  
  // one-sided difference gradient of u
  if (idx < Xsize-1 && idy < Ysize-1) {
    gradx[threadIdx.x+1][threadIdx.y+1] = u[threadIdx.x+2][threadIdx.y+1]-u[threadIdx.x+1][threadIdx.y+1];
    grady[threadIdx.x+1][threadIdx.y+1] = u[threadIdx.x+1][threadIdx.y+2]-u[threadIdx.x+1][threadIdx.y+1];  
  }

  // gradient values for ghost cells
  if (blockIdx.x>0 && threadIdx.x == 0){ 
    gradx[0][threadIdx.y+1] = u[1][threadIdx.y+1]-u[0][threadIdx.y+1];
    grady[0][threadIdx.y+1] = u[0][threadIdx.y+2]-u[0][threadIdx.y+1];
  }
  if (blockIdx.y > 0 && threadIdx.y == 0){
    gradx[threadIdx.x+1][0] = u[threadIdx.x+2][0] - u[threadIdx.x+1][0];
    grady[threadIdx.x+1][0] = u[threadIdx.x+1][1] - u[threadIdx.x+1][0];
  } 
  __syncthreads();

  // gradient descent step
  if (idx < Xsize && idy < Ysize) {
    numx = px[idx*Ysize+idy] + (tau/lambda)*gradx[threadIdx.x+1][threadIdx.y+1];
    numy = py[idx*Ysize+idy] + (tau/lambda)*grady[threadIdx.x+1][threadIdx.y+1];
    norm = sqrt( numx*numx + numy*numy);
    pxsh[threadIdx.x+1][threadIdx.y+1] = numx/max(1.0,norm);
    pysh[threadIdx.x+1][threadIdx.y+1] = numy/max(1.0,norm);
  }
  // gradient descent step for ghost cells 
  if(blockIdx.x > 0 && threadIdx.x == 0 ){
    numx = px[(idx-1)*Ysize+idy] + (tau/lambda)*gradx[0][threadIdx.y+1];
    numy = py[(idx-1)*Ysize+idy] + (tau/lambda)*grady[0][threadIdx.y+1];
    norm = sqrt( numx*numx + numy*numy);
    pxsh[0][threadIdx.y+1] = numx/max(1.0,norm);;
    pysh[0][threadIdx.y+1] = numy/max(1.0,norm);;
  }
  if(blockIdx.y > 0 && threadIdx.y == 0){
    numx = px[idx*Ysize+idy-1] + (tau/lambda)*gradx[threadIdx.x+1][0];
    numy = py[idx*Ysize+idy-1] + (tau/lambda)*grady[threadIdx.x+1][0];
    norm = sqrt( numx*numx + numy*numy);
    pxsh[threadIdx.x+1][0] = numx/max(1.0,norm);
    pysh[threadIdx.x+1][0] = numy/max(1.0,norm);
  }
  __syncthreads();
  float ux;
  float uy;
  // divergence values ux, uy (derivatives of p) for boundaries
  if ( idx == 0) {
    ux = pxsh[threadIdx.x+1][threadIdx.y+1]; //p1x[idx*Ysize + idy];
  }  
  if (idx == Xsize -1 ) {
    ux = -pxsh[threadIdx.x][threadIdx.y+1]; //-p1x[(idx-1)*Ysize + idy];
  }
  if (idy == 0){
    uy = pysh[threadIdx.x+1][threadIdx.y+1]; //p1y[idx*Ysize + idy];
  }
  if (idy == Ysize-1){
    uy = -pysh[threadIdx.x+1][threadIdx.y]; //-p1y[idx*Ysize + idy-1];
  }
  // div p at inner grid points
  if (idx > 0 && idx < Xsize -1) {
    ux = pxsh[threadIdx.x+1][threadIdx.y+1]-pxsh[threadIdx.x][threadIdx.y+1];// p1x[idx*Ysize + idy] - p1x[(idx-1)*Ysize + idy];
  }
  if (idy > 0 && idy < Ysize -1) {
    uy = pysh[threadIdx.x+1][threadIdx.y+1]-pysh[threadIdx.x+1][threadIdx.y]; //p1y[idx*Ysize + idy] - p1y[idx*Ysize + idy-1];
  }
  // divergence of p
  if (idx < Xsize && idy < Ysize) {
    div[idx*Ysize + idy] = ux + uy;
    px[idx*Ysize + idy] = pxsh[threadIdx.x+1][threadIdx.y+1]; //p1x[idx*Ysize + idy];
    py[idx*Ysize + idy] = pysh[threadIdx.x+1][threadIdx.y+1]; //p1y[idx*Ysize + idy];
  }
}

/*
u = f + \lambda div p
*/
__global__ void compute_u(float* u, float *f, float lambda, long Xsize, long Ysize, float* div){
  int idx = (blockIdx.x)*BLOCK_DIM + threadIdx.x;
  int idy = (blockIdx.y)*BLOCK_DIM + threadIdx.y;
  if (idx < Xsize && idy < Ysize) {
    u[idx*Ysize+idy] = f[idx*Ysize+idy] + lambda*div[idx*Ysize+idy];
  }
}

int main(int argc, char * argv[] ) {
  long T = 250; 
  float lambda = 100; 
  float mu = 0;
  float sigma = 50;
  float tau = 0.245;
  const char fname[] = "car.ppm";
  
  //sscanf(argv[1],"%d",&T);
  //sscanf(argv[2],"%d",&N);
  //sscanf(argv[2],"%f",&lambda);
  //sscanf(argv[2],"%f",&sigma);
   
  // Load image from file
  RGBImage u0, unoise;
  read_image(fname, &u0);
 
  long Xsize = u0.Xsize;
  long Ysize = u0.Ysize;
  unoise.Xsize = Xsize;
  unoise.Ysize = Ysize;
  float h = 1.0;
  unoise.A = (float*) malloc(3*Xsize*Ysize*sizeof(float));  
 
  // add noise to clean image 
  for(int c = 0; c < 3; c++){
    for(int i = 0; i < Xsize; i++){
      for(int j = 0; j < Ysize; j++) {
        unoise.A[c*Xsize*Ysize+ i*Ysize + j] = u0.A[c*Xsize*Ysize + i*Ysize + j] + randn(mu,sigma);
      }
    }
  }

  // write noisy image
  write_image("rof-noise.ppm",unoise);
 
  Timer t;
  // Allocate GPU memory
  float *ugpu, *fgpu, *p1xgpu, *p1ygpu, *p0xgpu, *p0ygpu, *gradx, *grady, *div;
  hipMalloc(&ugpu, 3*Xsize*Ysize*sizeof(float));
  hipMalloc(&fgpu, 3*Xsize*Ysize*sizeof(float));
  hipMalloc(&p1xgpu, 3*Xsize*Ysize*sizeof(float));
  hipMalloc(&p1ygpu, 3*Xsize*Ysize*sizeof(float));
  hipMalloc(&p0xgpu, 3*Xsize*Ysize*sizeof(float));
  hipMalloc(&p0ygpu, 3*Xsize*Ysize*sizeof(float));
  hipMalloc(&gradx, 3*Xsize*Ysize*sizeof(float));
  hipMalloc(&grady, 3*Xsize*Ysize*sizeof(float));
  hipMalloc(&div, 3*Xsize*Ysize*sizeof(float));

 
  hipMemcpy(ugpu, unoise.A, 3*Xsize*Ysize*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(fgpu, unoise.A, 3*Xsize*Ysize*sizeof(float), hipMemcpyHostToDevice);

  // Create streams
  hipStream_t streams[3];
  hipStreamCreate(&streams[0]);
  hipStreamCreate(&streams[1]);
  hipStreamCreate(&streams[2]);

  dim3 blockDim(BLOCK_DIM, BLOCK_DIM);
  dim3 gridDim(Xsize/BLOCK_DIM+1, Ysize/BLOCK_DIM+1);
  
 
  hipDeviceSynchronize();
  t.tic();

  for (long n = 0; n < T; n++) {
    rof<<<gridDim,blockDim, 0, streams[0]>>>(ugpu+0*Xsize*Ysize, p0xgpu+0*Xsize*Ysize, p1xgpu+0*Xsize*Ysize, p0ygpu+0*Xsize*Ysize, p1ygpu+0*Xsize*Ysize, fgpu+0*Xsize*Ysize, gradx+0*Xsize*Ysize, grady+0*Xsize*Ysize, lambda, tau, Xsize, Ysize, h, div+0*Xsize*Ysize);
    rof<<<gridDim,blockDim, 1, streams[1]>>>(ugpu+1*Xsize*Ysize, p0xgpu+1*Xsize*Ysize, p1xgpu+1*Xsize*Ysize, p0ygpu+1*Xsize*Ysize, p1ygpu+1*Xsize*Ysize, fgpu+1*Xsize*Ysize, gradx+1*Xsize*Ysize, grady+1*Xsize*Ysize, lambda, tau, Xsize, Ysize, h, div+1*Xsize*Ysize);
    rof<<<gridDim,blockDim, 2, streams[2]>>>(ugpu+2*Xsize*Ysize, p0xgpu+2*Xsize*Ysize, p1xgpu+2*Xsize*Ysize, p0ygpu+2*Xsize*Ysize, p1ygpu+2*Xsize*Ysize, fgpu+2*Xsize*Ysize, gradx+2*Xsize*Ysize, grady+2*Xsize*Ysize, lambda, tau, Xsize, Ysize, h, div+2*Xsize*Ysize);

  }

  hipDeviceSynchronize();
  double tt = t.toc();
  printf("nsmem GPU time = %fs\n", tt);
  printf("GPU Bandwidth = %f GB/s\n", (24*Xsize*Ysize*T*sizeof(float)+3*Xsize*Ysize*sizeof(float))/tt/1e9);
  hipMemcpy(u0.A, ugpu, 3*Xsize*Ysize*sizeof(float), hipMemcpyDeviceToHost);
 
  // Write output
  write_image("rof-nsmem.ppm", u0);
 
  hipFree(gradx);
  hipFree(grady);
  hipFree(p1xgpu);
  hipFree(p1ygpu);
  hipFree(p0xgpu);
  hipFree(p0ygpu);
  hipFree(div);
  
  hipMalloc(&p0xgpu, 3*Xsize*Ysize*sizeof(float));
  hipMalloc(&p0ygpu, 3*Xsize*Ysize*sizeof(float));
  hipMalloc(&div, 3*Xsize*Ysize*sizeof(float));

  hipDeviceSynchronize();
  t.tic();

  for (long n = 0; n < T; n++) {
    rof_gsmem<<<gridDim,blockDim, 0, streams[0]>>>(p0xgpu+0*Xsize*Ysize, p0ygpu+0*Xsize*Ysize, fgpu+0*Xsize*Ysize, lambda, tau, Xsize, Ysize, div+0*Xsize*Ysize);
    rof_gsmem<<<gridDim,blockDim, 1, streams[1]>>>(p0xgpu+1*Xsize*Ysize, p0ygpu+1*Xsize*Ysize, fgpu+1*Xsize*Ysize, lambda, tau, Xsize, Ysize, div+1*Xsize*Ysize);
    rof_gsmem<<<gridDim,blockDim, 2, streams[2]>>>(p0xgpu+2*Xsize*Ysize, p0ygpu+2*Xsize*Ysize, fgpu+2*Xsize*Ysize, lambda, tau, Xsize, Ysize, div+2*Xsize*Ysize);
  }

  compute_u<<<gridDim,blockDim, 0, streams[0]>>>(ugpu+0*Xsize*Ysize, fgpu+0*Xsize*Ysize, lambda, Xsize, Ysize, div+0*Xsize*Ysize);
  compute_u<<<gridDim,blockDim, 1, streams[1]>>>(ugpu+1*Xsize*Ysize, fgpu+1*Xsize*Ysize, lambda, Xsize, Ysize, div+1*Xsize*Ysize);
  compute_u<<<gridDim,blockDim, 2, streams[2]>>>(ugpu+2*Xsize*Ysize, fgpu+2*Xsize*Ysize, lambda, Xsize, Ysize, div+2*Xsize*Ysize);

  hipDeviceSynchronize();
  tt = t.toc();
  printf("gsmem GPU time = %fs\n", tt);
  printf("GPU Bandwidth = %f GB/s\n", (7*Xsize*Ysize*T + 6*Xsize*Ysize*T/BLOCK_DIM+3*Xsize*Ysize)*sizeof(float)/tt/1e9);
  hipMemcpy(u0.A, ugpu, 3*Xsize*Ysize*sizeof(float), hipMemcpyDeviceToHost);

  // Write output
  write_image("rof-gsmem.ppm", u0);


  // Free memory
  hipStreamDestroy(streams[0]);
  hipStreamDestroy(streams[1]);
  hipStreamDestroy(streams[2]);
  hipFree(ugpu);
  hipFree(fgpu);
  hipFree(div);
  hipFree(p0xgpu);
  hipFree(p0ygpu);
  free_image(&u0);
  free_image(&unoise);
  return 0;
}

